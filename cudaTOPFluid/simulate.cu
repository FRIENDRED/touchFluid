/* 
	Simulation functions that kick off CUDA kernels
	Kurt Kaminski 2016
*/

#include "kernels.cuh"
#include "common.cuh"

dim3 grid, threads;

bool runOnce = true;
int dimX, dimY, size;
float *chemA, *chemA_prev, *chemB, *chemB_prev, *laplacian;
float *vel[2], *vel_prev[2];
float *pressure, *pressure_prev;
float *temperature, *temperature_prev;
float *density, *density_prev;
float *divergence;
float *boundary;

// incoming data
float *mouse, *mouse_old;
const TCUDA_ParamInfo *mouseCHOP;
const TCUDA_ParamInfo *boundaryTOP;
const TCUDA_ParamInfo *F_TOP;
const TCUDA_ParamInfo *rdCHOP;

float dt = .1;
float diff = 0.00001f;
float visc = 0.000001f;
float force = 30.;
float buoy = 0.0;
float source_density = 2.0;
float source_temp = .25;
float dA = 0.0002; // diffusion constants
float dB = 0.00001;

char* TCUDA_DataType_enum[];
char* TCUDA_ProjectionType_enum[];
char* TCUDA_ObjSubType_enum[];
char* TCUDA_DataFormat_enum[];
char* TCUDA_ParamDataFormat_enum[];
char* TCUDA_DataLocation_enum[];
char* TCUDA_ChanOrder_enum[];
char* TCUDA_OutputType_enum[];
char* TCUDA_PixelFormat_enum[];
char* TCUDA_ObjParamType_enum[];
char* TCUDA_FogType_enum[];
char* TCUDA_MemType_enum[];

// ffmpeg -i [input] -c:v libvpx -b:v 1M [output].webm
bool hasEnding (std::string const &fullString, std::string const &ending) {
    if (fullString.length() >= ending.length()) {
        return (0 == fullString.compare (fullString.length() - ending.length(), ending.length(), ending));
    } else {
        return false;
    }
}

bool hasBeginning (std::string const &fullString, std::string const &beginning) {
	if (fullString.find(beginning) != std::string::npos )
		return true;
	else 
		return false;
}

///////////////////////////////////////////////////////////////////////////////
// Print connected node information
///////////////////////////////////////////////////////////////////////////////
void printNodeInfo(const int nparams, const TCUDA_ParamInfo **params){
	printf("\n----------\nINCOMING PARAMETERS:\n");
	printf("%d nodes connected\n\n", nparams);

	for (int i = 0; i < nparams; i++) {
		printf("Node %d: %s\n", params[i]->inputNumber, params[i]->name);
		printf("%d values\n", params[i]->param.numValues);
		if (params[i]->dataType == TCUDA_DATA_TYPE_TOP){
			printf("  TOP INFO:\n");
			printf("  w: %d, h: %d\n", params[i]->top.width, params[i]->top.height);
			printf("  %s\n", TCUDA_ChanOrder_enum[params[i]->top.chanOrder]); 
		}
		if (params[i]->dataType == TCUDA_DATA_TYPE_CHOP){
			printf("  CHOP INFO:\n");
			printf("  Num channels: %d\n", params[i]->chop.numChannels); 
			printf("  Length: %d\n", params[i]->chop.length);
			printf("  Sample rate: %f\n", params[i]->chop.sampleRate); 
		}
		printf("\n");
	}
	printf("----------\n\n");
}

///////////////////////////////////////////////////////////////////////////////
// Find CHOPS
///////////////////////////////////////////////////////////////////////////////
void findCHOPS(const int nparams, const TCUDA_ParamInfo **params){
	for (int i = 0; i < nparams; i++){
		if (hasBeginning(params[i]->name, "OUT_mouse")){
			mouseCHOP = params[i];
			printf("findCHOPS(): found mouse: %s\n", mouseCHOP->name);
		}
		if (hasBeginning(params[i]->name, "OUT_boundary")){
			boundaryTOP = params[i];
			printf("findCHOPS(): found Boundary TOP: %s\n", boundaryTOP->name);
		}
		if (hasBeginning(params[i]->name, "OUT_rdF")){
			F_TOP = params[i];
			printf("findCHOPS(): found F TOP: %s\n", F_TOP->name);
		}
		if (hasBeginning(params[i]->name, "OUT_rdCHOP")){
			rdCHOP = params[i];
			printf("findCHOPS(): found rd CHOP: %s\n", rdCHOP->name);
		}
	}

}

///////////////////////////////////////////////////////////////////////////////
// Set up global variables
///////////////////////////////////////////////////////////////////////////////
void initVariables(const TCUDA_ParamInfo **_params, const TCUDA_ParamInfo *_output)
{
	// Set container dimensions to whatever the incoming TOP is set to
	dimX = _output->top.width;
	dimY = _output->top.height;
	size = dimX * dimY;

	threads = dim3(16,16);
	grid.x = (dimX + threads.x - 1) / threads.x;
	grid.y = (dimY + threads.y - 1) / threads.y;
	
	printf("-- DIMENSIONS: %d x %d --\n", dimX, dimY);
	
	// Get mouse info
	int num_mouse_chans = mouseCHOP->chop.numChannels;
	mouse = (float*)malloc(sizeof(float)*num_mouse_chans);
	mouse_old = (float*)malloc(sizeof(float)*num_mouse_chans);
	hipMemcpy(mouse, (float*)mouseCHOP->data, sizeof(float)*num_mouse_chans, hipMemcpyDeviceToHost);
	for (int i=0; i<num_mouse_chans; i++){
		mouse_old[i]=mouse[i];
	}
	printf("initVariables(): done.\n");
}

///////////////////////////////////////////////////////////////////////////////
// Allocate GPU memory
///////////////////////////////////////////////////////////////////////////////
void initCUDA() 
{
	hipMalloc((void**)&chemA, sizeof(float)*size);
	hipMalloc((void**)&chemA_prev, sizeof(float)*size);
	hipMalloc((void**)&chemB, sizeof(float)*size);
	hipMalloc((void**)&chemB_prev, sizeof(float)*size);
	hipMalloc((void**)&laplacian, sizeof(float)*size);
	hipMalloc((void**)&boundary, sizeof(float)*size * 4);

	for (int i=0; i<2; i++){
		hipMalloc((void**)&vel[i], sizeof(int)*size);
		hipMalloc((void**)&vel_prev[i], sizeof(int)*size);
	}

	hipMalloc((void**)&pressure, sizeof(float)*size );
	hipMalloc((void**)&pressure_prev, sizeof(float)*size );
	hipMalloc((void**)&temperature, sizeof(float)*size );
	hipMalloc((void**)&temperature_prev, sizeof(float)*size );
	hipMalloc((void**)&density, sizeof(float)*size );
	hipMalloc((void**)&density_prev, sizeof(float)*size );
	hipMalloc((void**)&divergence, sizeof(float)*size );

	printf("initCUDA(): Allocated GPU memory.\n");
}

///////////////////////////////////////////////////////////////////////////////
// Initialize GPU arrays
///////////////////////////////////////////////////////////////////////////////
void initArrays() 
{
  for (int i=0; i<2; i++){
	  ClearArray<<<grid,threads>>>(vel[i], 0.0, dimX, dimY);
	  ClearArray<<<grid,threads>>>(vel_prev[i], 0.0, dimX, dimY);
  }

  ClearArray<<<grid,threads>>>(chemA, 1.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(chemA_prev, 1.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(chemB, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(chemB_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(laplacian, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(boundary, 0.0, dimX, dimY);

  ClearArray<<<grid,threads>>>(pressure, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(pressure_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(temperature, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(temperature_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(density, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(density_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(divergence, 0.0, dimX, dimY);

  printf("initArrays(): Initialized GPU arrays.\n");
}

///////////////////////////////////////////////////////////////////////////////
// Initialize
///////////////////////////////////////////////////////////////////////////////
void initialize(const int _nparams, const TCUDA_ParamInfo **_params, const TCUDA_ParamInfo *_output)
{
	printNodeInfo(_nparams, _params);
	findCHOPS(_nparams, _params);
	initVariables(_params, _output);
	initCUDA();
	initArrays();
	printf("initialize(): done.\n");
}

///////////////////////////////////////////////////////////////////////////////
// Get from UI
///////////////////////////////////////////////////////////////////////////////
void get_from_UI(const TCUDA_ParamInfo **params, float *_temp, float *_dens, float *_u, float *_v) 
{
	ClearArray<<<grid,threads>>>(chemA_prev, 1.0, dimX, dimY);
	ClearArray<<<grid,threads>>>(chemB_prev, 0.0, dimX, dimY);
	//ClearArray<<<grid,threads>>>(_u, 0.0, dimX, dimY);
	//ClearArray<<<grid,threads>>>(_v, 0.0, dimX, dimY);

	//DrawSquare<<<grid,threads>>>(_chemB0, 1.0, dimX, dimY);

	// Use first input as material for chemB
	//MakeSource<<<grid,threads>>>((int*)params[0]->data, _chemB0, dimX, dimY);
	
	// Use second input as boundary conditions
	//boundary = (float*)boundaryTOP->data;
	//MakeSource<<<grid,threads>>>((float*)boundaryTOP->data, boundary, dimX, dimY);
	
	// Apply obstacle velocity
	GetFromUI<<<grid,threads>>>(_u, _v, (float*)boundaryTOP->data, dimX, dimY);
	
	// Update mouse info
	hipMemcpy(mouse, (float*)mouseCHOP->data, sizeof(float)*mouseCHOP->chop.numChannels, hipMemcpyDeviceToHost);
	
	if ( mouse[2] < 1.0 && mouse[3] < 1.0 ) return;

	// map mouse position to window size
	//float mouse[0] = (float)(mouse_x)/(float)win_x;
	//float mouse[1] = (float)(win_y-mouse_y)/(float)win_y;
	int i, j = dimX * dimY;
	i = (int)(mouse[0]*dimX-1);
	j = (int)(mouse[1]*dimY-1);

	float x_diff = mouse[0]-mouse_old[0];
	float y_diff = mouse[1]-mouse_old[1];
	//printf("%f, %f\n", x_diff, y_diff);

	if ( i<1 || i>dimX || j<1 || j>dimY ) return;

	if ( mouse[2] > 0.0 && mouse[3] > 0.0) {
		GetFromUI<<<grid,threads>>>(_u, x_diff * force, i, j, dimX, dimY);
		GetFromUI<<<grid,threads>>>(_v, y_diff * force, i, j, dimX, dimY);
	}

	if ( mouse[3] > 0.0) {
		GetFromUI<<<grid,threads>>>(_dens, source_density, i, j, dimX, dimY);
		GetFromUI<<<grid,threads>>>(_temp, source_temp, i, j, dimX, dimY);
		//GetFromUI<<<grid,threads>>>(_chemB0, source_density, i, j, dimX, dimY);
//		particleSystem.addParticles(mouse[0], mouse[1], 100, .04);
	}

	if ( mouse[4] > 0.0 || mouse[5] > 0.0) {
		printf("Mouse wheel is down!\n");
	}

	for (int i=0; i<6; i++){
		mouse_old[i]=mouse[i];
	}

	return;
}

///////////////////////////////////////////////////////////////////////////////
// Density step
///////////////////////////////////////////////////////////////////////////////
void dens_step (  float *_chemA, float *_chemA0, float *_chemB, float *_chemB0,
				  float *u, float *v, float *bounds, float dt )
{

	// Naive ARD-----------------------
	AddSource<<<grid,threads>>>(_chemB, _chemB0, dt, dimX, dimY);
	//AddSource<<<grid,threads>>>(_chemA, _chemA0, dt, dimX, dimY);
	_chemA0 = _chemA;
	_chemB0 = _chemB;
	for (int i = 0; i < 2; i++){
		Diffusion<<<grid,threads>>>(_chemA, laplacian, bounds, dA, dt, dimX, dimY);
		AddLaplacian<<<grid,threads>>>(_chemA, laplacian, dimX, dimY);
		SetBoundary<<<grid,threads>>>(0, _chemA, bounds, dimX, dimY);
		ClearArray<<<grid,threads>>>(laplacian, 0.0, dimX, dimY);

		Diffusion<<<grid,threads>>>(_chemB, laplacian, bounds, dB, dt, dimX, dimY);
		AddLaplacian<<<grid,threads>>>(_chemB, laplacian, dimX, dimY);
		SetBoundary<<<grid,threads>>>(0, chemB, bounds, dimX, dimY);
		ClearArray<<<grid,threads>>>(laplacian, 0.0, dimX, dimY);

		for (int j = 0; j < 2; j++){
		React<<<grid,threads>>>( _chemA, _chemB, (float*)F_TOP->data, (float*)rdCHOP->data, bounds, dt, dimX, dimY );
		
		}
	}

	SWAP ( _chemA0, _chemA );
	SWAP ( _chemB0, _chemB );

	// Density advection: chemB
	Advect<<<grid,threads>>>(vel_prev[0], vel_prev[1], _chemA0, bounds, _chemA,
							dt, 1.0, true, dimX, dimY);

	// Density advection: chemB
	Advect<<<grid,threads>>>(vel_prev[0], vel_prev[1], _chemB0, bounds, _chemB,
							dt, 1.0, true, dimX, dimY);
}


///////////////////////////////////////////////////////////////////////////////
// Simulate
///////////////////////////////////////////////////////////////////////////////
static void simulate(const TCUDA_ParamInfo **params, const TCUDA_ParamInfo *output){


	// Velocity advection
	Advect<<<grid,threads>>>(vel_prev[0], vel_prev[1], vel_prev[0], vel_prev[1],
								(float*)boundaryTOP->data, vel[0], vel[1], 
								dt, .9995, dimX, dimY);
	SWAP(vel_prev[0], vel[0]);
	SWAP(vel_prev[1], vel[1]);

	// Temperature advection
	Advect<<<grid,threads>>>(vel_prev[0], vel_prev[1], temperature_prev, (float*)boundaryTOP->data, temperature,
							dt, .99, false, dimX, dimY);
	SWAP(temperature_prev, temperature);

	// Vorticity Confinement
	vorticityConfinement<<<grid,threads>>>( vel[0], vel[1], vel_prev[0], vel_prev[1], 
											(float*)boundaryTOP->data, dt, dimX, dimY);
		
	float Tamb = 0.0;
	getSum<<<grid,threads>>>(temperature_prev, Tamb, dimX, dimY);
	Tamb /= float(dimX * dimY);
	ApplyBuoyancy<<<grid,threads>>>(vel_prev[0], vel_prev[1], temperature_prev, chemB_prev,
									vel[0], vel[1], Tamb, dt, dimX, dimY);
	SWAP(vel_prev[0], vel[0]);
	SWAP(vel_prev[1], vel[1]);

	// Apply impulses
	get_from_UI(params, temperature_prev, chemB_prev, vel_prev[0], vel_prev[1]);

	// Density advection
	dens_step( chemA, chemA_prev, chemB, chemB_prev, vel_prev[0], vel_prev[1], (float*)boundaryTOP->data, dt );

	// Compute divergence
	ComputeDivergence<<<grid,threads>>>( vel_prev[0], vel_prev[1], (float*)boundaryTOP->data, divergence, dimX, dimY );

	// Pressure solve
	ClearArray<<<grid,threads>>>(pressure_prev, 0.0, dimX, dimY);
	for (int i=0; i<30; i++){
		Jacobi<<<grid,threads>>>(pressure_prev, divergence, (float*)boundaryTOP->data, pressure, dimX, dimY);
		SWAP(pressure_prev, pressure);
	}

	// Subtract pressure gradient from velocity
	SubtractGradient<<<grid,threads>>>( vel_prev[0], vel_prev[1], pressure_prev, (float*)boundaryTOP->data, 
										vel[0], vel[1], dimX, dimY);
	SWAP(vel_prev[0], vel[0]);
	SWAP(vel_prev[1], vel[1]);


	MakeColor<<<grid,threads>>>(chemA, chemB, vel[0], vel[1], (float*)output->data, dimX, dimY);
	//MakeColor<<<grid,threads>>>(chemB, (float*)boundaryTOP->data, chemB, (float*)output->data, dimX, dimY);
	//(float*)boundaryTOP->data

}

extern "C"
{
	// The main function where you should execute your CUDA kernel(s).
	// nparams is the number of parameters passed into this function
	// and params is the array of those parameters
	// output contains information about the output you need to write
	// output.data is the array that you write out to (this will be turned into a TOP by Touch)

	// 3d texture idea: output different Z slices with each frame #, compiling them into a Texture3d TOP
	//					would have to change framerate to compensate for #of slices/fps 
	DLLEXPORT bool
	tCudaExecuteKernel(const TCUDA_NodeInfo *info, const int nparams, const TCUDA_ParamInfo **params, const TCUDA_ParamInfo *output)
	{
		if (runOnce) {
			initialize(nparams, params, output);
			runOnce = false;
		}

		simulate(params, output);

		return true;
	}

}