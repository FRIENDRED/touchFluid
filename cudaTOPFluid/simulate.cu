/* 
	Simulation functions that kick off CUDA kernels
	Kurt Kaminski 2016
*/

#include <iostream>
#include <map>

#include "common.cuh"
#include "kernels.cuh"
#include "util_functions.hpp"
#include "private/TCUDA_Types.h"

using namespace std;

dim3 grid, threads;

bool runOnce = true;
bool initialized = false;

int dimX, dimY, size;
float *chemA, *chemA_prev, *chemB, *chemB_prev, *laplacian;
float *vel[2], *vel_prev[2];
float *pressure, *pressure_prev;
float *temperature, *temperature_prev;
float *density, *density_prev;
float *divergence;
float *boundary;

// incoming data
map<string, const TCUDA_ParamInfo*> nodes;
float *mouse, *mouse_old;
float *constants;
const TCUDA_ParamInfo *mouseCHOP;
const TCUDA_ParamInfo *densityTOP;
const TCUDA_ParamInfo *boundaryTOP;
const TCUDA_ParamInfo *rdCHOP;
const TCUDA_ParamInfo *constCHOP;
const TCUDA_ParamInfo *resetCHOP;

float dt = 0.1f;
float dA = 0.0002; // gray-scott
float dB = 0.00001;
float xLen = 100.0f;
float yLen = 100.0f;
int nDiff = 2;
int nReact = 1;
int nJacobi = 30;

//float dt = .02f;
//float dA = 0.75; // barkley model
//float dB = 0.0;

float diff = 0.00001f;
float visc = 0.000001f;
float force = 30.;
float buoy = 0.0;
float source_density = 2.0;
float source_temp = .25;

// ffmpeg -i [input] -c:v libvpx -b:v 1M [output].webm
// ffmpeg -i [input] -c:v libx264 -b:v 1M [output].webm

///////////////////////////////////////////////////////////////////////////////
// Find connected nodes for easier reference in a map
///////////////////////////////////////////////////////////////////////////////
bool findNodes(const int nparams, const TCUDA_ParamInfo **params){

	// fill nodes<> with key/value pairs
	nodes["mouse"] = mouseCHOP;
	nodes["density"] = densityTOP;
	nodes["boundary"] = boundaryTOP;
	nodes["rdCHOP"] = rdCHOP;
	nodes["constants"] = rdCHOP;
	nodes["reset"] = resetCHOP;


	// search incoming params[] for matching name and assigne nodes<> value to it
	bool missingNodes = false;
	typedef map<string, const TCUDA_ParamInfo*>::iterator iterator;
	for (iterator it = nodes.begin(); it != nodes.end(); it++) {
		for (int i = 0; i < nparams; i++){
			if (hasBeginning(params[i]->name, it->first.c_str())) {
				it->second = params[i];
				printf("findNodes(): found %s: %s\n", it->first.c_str(), it->second->name);
				break;
			}
			if (i == nparams-1) {
				printf("findNodes(): error: could not find %s!\n", it->first.c_str());
				missingNodes = true;
			}
		}
	}

	if (missingNodes) {
		printf("findNodes(): couldn't find required nodes. To continue, attach required nodes.\n");
		return false;
	}
	else {
		printf("findNodes(): done finding nodes.\n");
		return true;
	}

}

void getConstants() {
	// constants[] = {dt, xLen, yLen, nDiff, nReact, nJacobi}
	dt = constants[0];
	xLen = constants[1];
	yLen = constants[2];
	nDiff = (int)constants[3];
	nReact = (int)constants[4];
	nJacobi = (int)constants[5];
}

///////////////////////////////////////////////////////////////////////////////
// Set up global variables
///////////////////////////////////////////////////////////////////////////////
void initVariables(const TCUDA_ParamInfo **_params, const TCUDA_ParamInfo *_output)
{
	// Set container dimensions to whatever the incoming TOP is set to
	dimX = _output->top.width;
	dimY = _output->top.height;
	size = dimX * dimY;

	threads = dim3(16,16);
	grid.x = (dimX + threads.x - 1) / threads.x;
	grid.y = (dimY + threads.y - 1) / threads.y;
	
	printf("-- DIMENSIONS: %d x %d --\n", dimX, dimY);
	
	// Allocate mouse array
	mouse = (float*)malloc(sizeof(float)*nodes["mouse"]->chop.numChannels);
	mouse_old = (float*)malloc(sizeof(float)*nodes["mouse"]->chop.numChannels);
	
	// Local mouse pointer points to CHOP node
	mouse = (float*)nodes["mouse"]->data;
	for (int i = 0; i < nodes["mouse"]->chop.numChannels; i++){
		mouse_old[0]=mouse[1];
	}

	// Allocate constants array
	constants = (float*)malloc(sizeof(float)*nodes["constants"]->chop.numChannels);
	
	// Local constants pointer points to CHOP node
	constants = (float*)nodes["constants"]->data;
	getConstants();

	printf("initVariables(): done.\n");
}

///////////////////////////////////////////////////////////////////////////////
// Allocate GPU memory
///////////////////////////////////////////////////////////////////////////////
void initCUDA() 
{
	hipMalloc((void**)&chemA, sizeof(float)*size);
	hipMalloc((void**)&chemA_prev, sizeof(float)*size);
	hipMalloc((void**)&chemB, sizeof(float)*size);
	hipMalloc((void**)&chemB_prev, sizeof(float)*size);
	hipMalloc((void**)&laplacian, sizeof(float)*size);
	hipMalloc((void**)&boundary, sizeof(float)*size * 4);

	for (int i=0; i<2; i++){
		hipMalloc((void**)&vel[i], sizeof(int)*size);
		hipMalloc((void**)&vel_prev[i], sizeof(int)*size);
	}

	hipMalloc((void**)&pressure, sizeof(float)*size );
	hipMalloc((void**)&pressure_prev, sizeof(float)*size );
	hipMalloc((void**)&temperature, sizeof(float)*size );
	hipMalloc((void**)&temperature_prev, sizeof(float)*size );
	hipMalloc((void**)&density, sizeof(float)*size );
	hipMalloc((void**)&density_prev, sizeof(float)*size );
	hipMalloc((void**)&divergence, sizeof(float)*size );

	printf("initCUDA(): Allocated GPU memory.\n");
}

///////////////////////////////////////////////////////////////////////////////
// Initialize GPU arrays
///////////////////////////////////////////////////////////////////////////////
void initArrays() 
{
  for (int i=0; i<2; i++){
	  ClearArray<<<grid,threads>>>(vel[i], 0.0, dimX, dimY);
	  ClearArray<<<grid,threads>>>(vel_prev[i], 0.0, dimX, dimY);
  }

  ClearArray<<<grid,threads>>>(chemA, 1.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(chemA_prev, 1.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(chemB, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(chemB_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(laplacian, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(boundary, 0.0, dimX, dimY);

  ClearArray<<<grid,threads>>>(pressure, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(pressure_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(temperature, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(temperature_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(density, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(density_prev, 0.0, dimX, dimY);
  ClearArray<<<grid,threads>>>(divergence, 0.0, dimX, dimY);

  printf("initArrays(): Initialized GPU arrays.\n");
}

///////////////////////////////////////////////////////////////////////////////
// Initialize
///////////////////////////////////////////////////////////////////////////////
bool init(const int _nparams, const TCUDA_ParamInfo **_params, const TCUDA_ParamInfo *_output)
{
	printNodeInfo(_nparams, _params);

	if ( findNodes(_nparams, _params) ) {
		initVariables(_params, _output);
		initCUDA();
		initArrays();
		printf("init(): done.\n");
		return true;
	}
	else {
		printf("init(): could not initalize. Not simulating.\n");
		return false;
	}
}

///////////////////////////////////////////////////////////////////////////////
// Get from UI
///////////////////////////////////////////////////////////////////////////////
void get_from_UI(const TCUDA_ParamInfo **params, float *_temp, float *_dens, float *_u, float *_v) 
{
	//ClearArray<<<grid,threads>>>(chemA_prev, 1.0, dimX, dimY);
	ClearArray<<<grid,threads>>>(chemA_prev, 0.0, dimX, dimY);
	ClearArray<<<grid,threads>>>(chemB_prev, 0.0, dimX, dimY);
	//ClearArray<<<grid,threads>>>(_u, 0.0, dimX, dimY);
	//ClearArray<<<grid,threads>>>(_v, 0.0, dimX, dimY);

	//DrawSquare<<<grid,threads>>>(chemA, 1.0, dimX, dimY);
	//DrawSquare<<<grid,threads>>>(chemB, 0.75 * .5, dimX, dimY);
	
	// Apply incoming density
	SetFromUI<<<grid,threads>>>(chemA, chemB, (float*)nodes["density"]->data, dimX, dimY);
	
	// Apply obstacle velocity
	AddFromUI<<<grid,threads>>>(_u, _v, (float*)nodes["boundary"]->data, dimX, dimY);
	
	// Update mouse and constants info
	// mouse[] = {x, y, LMB, RMB, MMB, wheel}
	//mouse = (float*)nodes["mouse"]->data;
	getConstants();

	if ( mouse[2] < 1.0 && mouse[3] < 1.0 ) return;

	int i, j = dimX * dimY;
	i = (int)(mouse[0]*dimX-1);
	j = (int)(mouse[1]*dimY-1);

	float x_diff = mouse[0]-mouse_old[0];
	float y_diff = mouse[1]-mouse_old[1];
	//printf("%f, %f\n", x_diff, y_diff);

	if (i<1 || i>dimX || j<1 || j>dimY ) return;

	if (mouse[2] > 0.0 && mouse[3] > 0.0) {
		AddFromUI<<<grid,threads>>>(_u, x_diff * force, i, j, dimX, dimY);
		AddFromUI<<<grid,threads>>>(_v, y_diff * force, i, j, dimX, dimY);
	}

	if (mouse[3] > 0.0) {
		AddFromUI<<<grid,threads>>>(_dens, source_density, i, j, dimX, dimY);
		AddFromUI<<<grid,threads>>>(_temp, source_temp, i, j, dimX, dimY);
		//GetFromUI<<<grid,threads>>>(_chemB0, source_density, i, j, dimX, dimY);
		//particleSystem.addParticles(mouse[0], mouse[1], 100, .04);
	}

	if (mouse[4] > 0.0) printf("mouse[4] is down!\n");

	for (int i=0; i<6; i++){
		mouse_old[i]=mouse[i];
	}

	return;
}

///////////////////////////////////////////////////////////////////////////////
// Density step
///////////////////////////////////////////////////////////////////////////////
void dens_step (  float *_chemA, float *_chemA0, float *_chemB, float *_chemB0,
				  float *u, float *v, float *bounds, float dt )
{

	// Naive ARD-----------------------
	//AddSource<<<grid,threads>>>(_chemB, _chemB0, dt, dimX, dimY);
	//AddSource<<<grid,threads>>>(_chemA, _chemA0, dt, dimX, dimY);
	_chemA0 = _chemA;
	_chemB0 = _chemB;
	for (int i = 0; i < nDiff; i++){
		Diffusion<<<grid,threads>>>(_chemA, laplacian, bounds, dA, xLen, yLen, dt, dimX, dimY);
		AddLaplacian<<<grid,threads>>>(_chemA, laplacian, dimX, dimY);
		SetBoundary<<<grid,threads>>>(0, _chemA, bounds, dimX, dimY);
		ClearArray<<<grid,threads>>>(laplacian, 0.0, dimX, dimY);

		Diffusion<<<grid,threads>>>(_chemB, laplacian, bounds, dB, xLen, yLen, dt, dimX, dimY);
		AddLaplacian<<<grid,threads>>>(_chemB, laplacian, dimX, dimY);
		SetBoundary<<<grid,threads>>>(0, chemB, bounds, dimX, dimY);
		ClearArray<<<grid,threads>>>(laplacian, 0.0, dimX, dimY);

		for (int j = 0; j < nReact; j++){
			React<<<grid,threads>>>( _chemA, _chemB, (float*)nodes["rdCHOP"]->data, bounds, dt, dimX, dimY );
		}
	}

	SWAP ( _chemA0, _chemA );
	SWAP ( _chemB0, _chemB );

	// Density advection: chemB
	Advect<<<grid,threads>>>(vel_prev[0], vel_prev[1], _chemA0, bounds, _chemA,
							dt, 1.0, true, dimX, dimY);

	// Density advection: chemB
	Advect<<<grid,threads>>>(vel_prev[0], vel_prev[1], _chemB0, bounds, _chemB,
							dt, 1.0, true, dimX, dimY);
}


///////////////////////////////////////////////////////////////////////////////
// Simulate
///////////////////////////////////////////////////////////////////////////////
static void simulate(const TCUDA_ParamInfo **params, const TCUDA_ParamInfo *output){


	// Velocity advection
	Advect<<<grid,threads>>>(vel_prev[0], vel_prev[1], vel_prev[0], vel_prev[1],
								(float*)nodes["boundary"]->data, vel[0], vel[1], 
								dt, .9995, dimX, dimY);
	SWAP(vel_prev[0], vel[0]);
	SWAP(vel_prev[1], vel[1]);

	// Temperature advection
	Advect<<<grid,threads>>>(vel_prev[0], vel_prev[1], temperature_prev, (float*)nodes["boundary"]->data, temperature,
							dt, .99, false, dimX, dimY);
	SWAP(temperature_prev, temperature);

	// Vorticity Confinement
	vorticityConfinement<<<grid,threads>>>( vel[0], vel[1], vel_prev[0], vel_prev[1], 
											(float*)nodes["boundary"]->data, dt, dimX, dimY);
		
	float Tamb = 0.0;
	getSum<<<grid,threads>>>(temperature_prev, Tamb, dimX, dimY);
	Tamb /= float(dimX * dimY);
	ApplyBuoyancy<<<grid,threads>>>(vel_prev[0], vel_prev[1], temperature_prev, chemB_prev,
									vel[0], vel[1], Tamb, dt, dimX, dimY);
	SWAP(vel_prev[0], vel[0]);
	SWAP(vel_prev[1], vel[1]);

	// Apply impulses
	get_from_UI(params, temperature_prev, chemB_prev, vel_prev[0], vel_prev[1]);

	// Reaction-Diffusion and Density advection
	dens_step( chemA, chemA_prev, chemB, chemB_prev, vel_prev[0], vel_prev[1], (float*)nodes["boundary"]->data, dt );

	// Compute divergence
	ComputeDivergence<<<grid,threads>>>( vel_prev[0], vel_prev[1], (float*)nodes["boundary"]->data, divergence, dimX, dimY );

	// Pressure solve
	ClearArray<<<grid,threads>>>(pressure_prev, 0.0, dimX, dimY);
	for (int i=0; i<nJacobi; i++){
		Jacobi<<<grid,threads>>>(pressure_prev, divergence, (float*)nodes["boundary"]->data, pressure, dimX, dimY);
		SWAP(pressure_prev, pressure);
	}

	// Subtract pressure gradient from velocity
	SubtractGradient<<<grid,threads>>>( vel_prev[0], vel_prev[1], pressure_prev, (float*)nodes["boundary"]->data, 
										vel[0], vel[1], dimX, dimY);
	SWAP(vel_prev[0], vel[0]);
	SWAP(vel_prev[1], vel[1]);


	MakeColor<<<grid,threads>>>(chemA, chemB, vel[0], vel[1], (float*)output->data, dimX, dimY);
	//MakeColor<<<grid,threads>>>(chemB, (float*)nodes["boundary"]->data, chemB, (float*)output->data, dimX, dimY);

}

extern "C"
{
	// The main function to execute CUDA kernel(s).
	// nparams is the number of parameters passed into this function
	// and params is the array of those parameters
	// output contains information about the output you need to write
	// output.data is the array to write out to (this will be turned into a TOP by Touch)

	// 3d texture idea: output different Z slices with each frame #, compiling them into a Texture3d TOP
	//					would have to change framerate to compensate for #of slices/fps 
	DLLEXPORT bool
	tCudaExecuteKernel(const TCUDA_NodeInfo *info, const int nparams, const TCUDA_ParamInfo **params, const TCUDA_ParamInfo *output)
	{
		if (runOnce) {
			initialized = init(nparams, params, output);
			runOnce = false;
		}

		if (initialized) {
			if (*(float*)nodes["reset"]->data > 0.0f) {
				initArrays();
			}

			simulate(params, output);
			return true;
		}

		else return false;

	}

}